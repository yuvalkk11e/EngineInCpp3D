#include "hip/hip_runtime.h"
#include "GpuKernels.cuh"

//This is where all of our kernels code is :

#define PI 3.14159265
#define PIr (180/PI) // I CHANGED FROM PI/180 TO THIS !

void cudaCheckErrors(std::string msg)
{
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",
			msg, hipGetErrorString(err),
			__FILE__, __LINE__);
		fprintf(stderr, "*** FAILED - ABORTING\n");
		exit(1);
	}
}

__global__ void RenderingKernel(VertexBuffer *output)
{
	//(*output).getValue(0);
}

void Launch_Rendering(VertexBuffer &c, FaceBuffer &a, int View_Angle ,int HEIGHT, int WIDTH, int View_Distance)
{
	Triangle **dev_a;
	VertexBuffer *dev_c;
	hipError_t cudaStatus;

	int size = a.getSize();

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);

	// Allocate GPU memory for our classes    .
	cudaStatus = hipMalloc((void**)&dev_a, sizeof(Triangle)*size);
	cudaCheckErrors("malloc2");

	cudaStatus = hipMalloc((void**)&dev_c, sizeof(VertexBuffer));
	cudaCheckErrors("malloc1");

	// copy our classes to allocated memory
	cudaStatus = hipMemcpy(dev_c, &c, sizeof(VertexBuffer), hipMemcpyHostToDevice);
	cudaCheckErrors("MEMCPY2");
	
	hipMemcpy(dev_a, a.getValues(), sizeof(Triangle)*size, hipMemcpyHostToDevice);


	//cudaStatus = hipMemcpy(dev_a, &a, sizeof(FaceBuffer), hipMemcpyHostToDevice);
	//cudaCheckErrors("MEMCPY1");

	/*Copy input vectors from host memory to GPU classes .
	Triangle * data;
	hipMalloc((void**)&data, sizeof(Triangle)*size);
	hipMemcpy(data, a.getValues(), sizeof(Triangle)*size, hipMemcpyHostToDevice);
	hipMemcpy(&(dev_c), &data, sizeof(Triangle *)*size, hipMemcpyHostToDevice); //cant access data of dev_c
	*/

	Vector3 up(0, 1, 0);
	float angle_bith = View_Angle / HEIGHT;
	float angle_bitw = View_Angle / WIDTH;
	// Launch a kernel on the GPU with one thread for each element.
	RenderingKernel << <size, 3>> > (dev_c);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
}